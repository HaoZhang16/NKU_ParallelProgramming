#include <queue>
#include <vector>
#include <cfloat>         // 添加FLT_MAX定义
#include <algorithm>      // 添加swap函数定义
#include <hip/hip_runtime.h>

#include <iostream>

// CUDA错误检查宏
#define CUDA_CHECK(call) \
do { \
    hipError_t err = (call); \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \"%s\"\n", \
        __FILE__, __LINE__, err, hipGetErrorString(err), #call); \
        exit(EXIT_FAILURE); \
    } \
} while (0)

// 设备交换函数
template <typename T>
__device__ void swap_device(T& a, T& b) {
    T t = a;
    a = b;
    b = t;
}

// 内核函数：每个线程块处理一个查询向量
__global__ void flat_batch_search_kernel_nolock(
    const float* base,            // 数据库向量 [n][d]
    const float* query,           // 查询向量 [m][d]
    size_t base_number,           // 数据库数量 n
    size_t query_number,          // 查询数量 m
    size_t vecdim,                // 维度 d
    size_t k,                     // top-k
    float* all_query_distances,   // 输出 [m][k]
    uint32_t* all_query_indices   // 输出 [m][k]
) {
    const size_t qid = blockIdx.x * blockDim.x + threadIdx.x;
    if (qid >= query_number) return;

    // 为每个线程分配自己的堆空间
    extern __shared__ char shared[];
    float* local_dist = new float[k];
    uint32_t* local_idx = new uint32_t[k];
    int heap_size = 0;

    const float* qvec = query + qid * vecdim;

    for (size_t i = 0; i < base_number; ++i) {
        const float* bvec = base + i * vecdim;
        float dot = 0.0f;
        for (size_t j = 0; j < vecdim; ++j)
            dot += bvec[j] * qvec[j];

        float dist = 1.0f - dot;

        if (heap_size < k) {
            // 插入
            int pos = heap_size++;
            local_dist[pos] = dist;
            local_idx[pos] = i;
            // 上浮
            while (pos > 0) {
                int parent = (pos - 1) / 2;
                if (local_dist[parent] < local_dist[pos]) {
                    swap_device(local_dist[parent], local_dist[pos]);
                    swap_device(local_idx[parent], local_idx[pos]);
                    pos = parent;
                } else break;
            }
        } else if (dist < local_dist[0]) {
            // 替换堆顶
            local_dist[0] = dist;
            local_idx[0] = i;
            // 下沉
            int pos = 0;
            while (true) {
                int left = 2 * pos + 1;
                int right = 2 * pos + 2;
                int largest = pos;
                if (left < k && local_dist[left] > local_dist[largest]) largest = left;
                if (right < k && local_dist[right] > local_dist[largest]) largest = right;
                if (largest != pos) {
                    swap_device(local_dist[pos], local_dist[largest]);
                    swap_device(local_idx[pos], local_idx[largest]);
                    pos = largest;
                } else break;
            }
        }
    }

    // 写回结果
    size_t offset = qid * k;
    for (size_t i = 0; i < k; ++i) {
        all_query_distances[offset + i] = (i < heap_size) ? local_dist[i] : FLT_MAX;
        all_query_indices[offset + i] = (i < heap_size) ? local_idx[i] : 0;
    }

    delete[] local_dist;
    delete[] local_idx;
}


std::vector<std::priority_queue<std::pair<float, uint32_t>>> flat_search_cuda(
    float* base,           // base[n][d]
    float* query,          // query[m][d]
    size_t base_number,    // n
    size_t query_number,   // m
    size_t vecdim,         // d
    size_t k               // 返回的top-k数量
) {
    // 设备指针
    float *d_base = nullptr, *d_query = nullptr;
    float *d_query_distances = nullptr;
    uint32_t *d_query_indices = nullptr;

    // 分配设备内存
    size_t base_size = base_number * vecdim * sizeof(float);
    size_t query_size = query_number * vecdim * sizeof(float);
    size_t results_dist_size = query_number * k * sizeof(float);
    size_t results_idx_size = query_number * k * sizeof(uint32_t);

    CUDA_CHECK(hipMalloc(&d_base, base_size));
    CUDA_CHECK(hipMalloc(&d_query, query_size));
    CUDA_CHECK(hipMalloc(&d_query_distances, results_dist_size));
    CUDA_CHECK(hipMalloc(&d_query_indices, results_idx_size));

    // 拷贝数据到设备
    CUDA_CHECK(hipMemcpy(d_base, base, base_size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_query, query, query_size, hipMemcpyHostToDevice));

    // 设置线程块和共享内存
    size_t threads_per_block = 64;
    size_t num_blocks = (query_number + threads_per_block - 1) / threads_per_block;

    // 启动改进核函数（不需要共享内存）
    flat_batch_search_kernel_nolock<<<num_blocks, threads_per_block>>>(
        d_base, d_query, base_number, query_number, vecdim, k,
        d_query_distances, d_query_indices
    );


    // 拷贝结果回主机
    std::vector<float> host_query_distances(query_number * k);
    std::vector<uint32_t> host_query_indices(query_number * k);
    CUDA_CHECK(hipMemcpy(host_query_distances.data(), d_query_distances, 
                         results_dist_size, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(host_query_indices.data(), d_query_indices,
                         results_idx_size, hipMemcpyDeviceToHost));

    // 释放设备内存
    CUDA_CHECK(hipFree(d_base));
    CUDA_CHECK(hipFree(d_query));
    CUDA_CHECK(hipFree(d_query_distances));
    CUDA_CHECK(hipFree(d_query_indices));

    // 构建结果优先级队列
    std::vector<std::priority_queue<std::pair<float, uint32_t>>> result(query_number);
    size_t valid_count = std::min(k, base_number); // 有效结果数量

    for (size_t qid = 0; qid < query_number; ++qid) {
        auto& pq = result[qid];
        size_t offset = qid * k;
        // 只添加有效结果
        for (size_t i = 0; i < valid_count; ++i) {
            size_t idx = offset + i;
            pq.emplace(host_query_distances[idx], host_query_indices[idx]);
        }
    }

    return result;
}


// ivf
__global__ void ivf_batch_search_kernel(
    const float* __restrict__ query,
    const float* __restrict__ new_base,
    const uint32_t* __restrict__ new_to_old,
    const uint32_t* __restrict__ cluster_start,
    const uint32_t* __restrict__ selected_clusters,
    size_t vecdim,
    size_t k,
    size_t m,
    float* out_distances,
    uint32_t* out_indices
) {
    extern __shared__ float shmem[]; // 动态共享内存
    float* sh_dist = shmem;
    uint32_t* sh_idx = (uint32_t*)(shmem + blockDim.x * k);

    int qid = blockIdx.x;
    int tid = threadIdx.x;
    const float* qvec = query + qid * vecdim;
    const uint32_t* clusters = selected_clusters + qid * m;

    // 1. 每个线程维护自己的局部top-k
    float local_dist[16];
    uint32_t local_idx[16];
    int local_size = 0;

    for (int mi = 0; mi < m; ++mi) {
        uint32_t cid = clusters[mi];
        uint32_t start = cluster_start[cid];
        uint32_t end = cluster_start[cid + 1];

        for (uint32_t j = start + tid; j < end; j += blockDim.x) {
            const float* base_ptr = new_base + j * vecdim;
            float dot = 0.0f;
            #pragma unroll
            for (int d = 0; d < vecdim; ++d) {
                dot += base_ptr[d] * qvec[d];
            }
            float dist = 1.0f - dot;

            if (local_size < k) {
                local_dist[local_size] = dist;
                local_idx[local_size] = new_to_old[j];
                local_size++;
            } else {
                int max_id = 0;
                for (int t = 1; t < k; ++t) {
                    if (local_dist[t] > local_dist[max_id]) max_id = t;
                }
                if (dist < local_dist[max_id]) {
                    local_dist[max_id] = dist;
                    local_idx[max_id] = new_to_old[j];
                }
            }
        }
    }

    // 2. 将局部结果写入共享内存
    for (int i = 0; i < k; ++i) {
        if (i < local_size) {
            sh_dist[tid * k + i] = local_dist[i];
            sh_idx[tid * k + i] = local_idx[i];
        } else {
            sh_dist[tid * k + i] = FLT_MAX; // 用最大值填充空位
            sh_idx[tid * k + i] = UINT32_MAX;
        }
    }
    __syncthreads();

    // 3. 线程0合并所有线程的结果
    if (tid == 0) {
        float final_dist[16];
        uint32_t final_idx[16];
        int final_size = 0;

        // 初始化最终结果
        for (int i = 0; i < k; ++i) {
            final_dist[i] = FLT_MAX;
        }

        // 遍历所有候选元素 (blockDim.x * k)
        for (int i = 0; i < blockDim.x * k; ++i) {
            float dist = sh_dist[i];
            if (dist == FLT_MAX) continue;

            if (final_size < k) {
                final_dist[final_size] = dist;
                final_idx[final_size] = sh_idx[i];
                final_size++;
            } else {
                // 查找当前结果中的最大距离
                int max_id = 0;
                for (int t = 1; t < k; ++t) {
                    if (final_dist[t] > final_dist[max_id]) max_id = t;
                }
                // 替换最大元素
                if (dist < final_dist[max_id]) {
                    final_dist[max_id] = dist;
                    final_idx[max_id] = sh_idx[i];
                }
            }
        }

        // 4. 写入全局内存
        float* dst_dist = out_distances + qid * k;
        uint32_t* dst_idx = out_indices + qid * k;
        for (int i = 0; i < k; ++i) {
            dst_dist[i] = (i < final_size) ? final_dist[i] : FLT_MAX;
            dst_idx[i] = (i < final_size) ? final_idx[i] : UINT32_MAX;
        }
    }
}

std::vector<std::priority_queue<std::pair<float, uint32_t>>> ivf_search_cuda(
    float* query,           // [batch][vecdim]
    float* centroids,       // [n_clusters][vecdim]
    float* new_base,        // [N][vecdim]
    uint32_t* new_to_old,   // [N]
    uint32_t* cluster_start,// [n_clusters + 1]
    size_t vecdim,       
    size_t k,             
    size_t n_clusters,     
    size_t m,               
    size_t batch_size     
) {
    std::vector<uint32_t> selected_clusters(batch_size * m);
    for (size_t qid = 0; qid < batch_size; ++qid) {
        std::vector<std::pair<float, uint32_t>> dist_id;
        for (size_t cid = 0; cid < n_clusters; ++cid) {
            float dot = 0.0f;
            for (size_t j = 0; j < vecdim; ++j) {
                dot += query[qid * vecdim + j] * centroids[cid * vecdim + j];
            }
            float dist = 1.0f - dot;
            dist_id.emplace_back(dist, cid);
        }
        std::partial_sort(dist_id.begin(), dist_id.begin() + m, dist_id.end());
        for (size_t i = 0; i < m; ++i) {
            selected_clusters[qid * m + i] = dist_id[i].second;
        }
    }

    // 分配显存
    float *d_query, *d_new_base, *d_out_dist;
    uint32_t *d_new_to_old, *d_cluster_start, *d_selected_clusters, *d_out_idx;
    size_t total_base = cluster_start[n_clusters];

    CUDA_CHECK(hipMalloc(&d_query, batch_size * vecdim * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_new_base, total_base * vecdim * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_new_to_old, total_base * sizeof(uint32_t)));
    CUDA_CHECK(hipMalloc(&d_cluster_start, (n_clusters + 1) * sizeof(uint32_t)));
    CUDA_CHECK(hipMalloc(&d_selected_clusters, batch_size * m * sizeof(uint32_t)));
    CUDA_CHECK(hipMalloc(&d_out_dist, batch_size * k * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_out_idx, batch_size * k * sizeof(uint32_t)));

    // 拷贝数据到GPU
    CUDA_CHECK(hipMemcpy(d_query, query, batch_size * vecdim * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_new_base, new_base, total_base * vecdim * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_new_to_old, new_to_old, total_base * sizeof(uint32_t), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_cluster_start, cluster_start, (n_clusters + 1) * sizeof(uint32_t), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_selected_clusters, selected_clusters.data(), batch_size * m * sizeof(uint32_t), hipMemcpyHostToDevice));

    // 启动核函数
    size_t threads = 128;
    size_t shared_mem_size = 2 * threads * k * sizeof(float); // 距离+索引
    ivf_batch_search_kernel<<<batch_size, threads, shared_mem_size>>>(
        d_query, d_new_base, d_new_to_old, d_cluster_start, d_selected_clusters,
        vecdim, k, m, d_out_dist, d_out_idx
    );
    CUDA_CHECK(hipDeviceSynchronize());

    // 拷回结果
    std::vector<float> host_dist(batch_size * k);
    std::vector<uint32_t> host_idx(batch_size * k);
    CUDA_CHECK(hipMemcpy(host_dist.data(), d_out_dist, batch_size * k * sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(host_idx.data(), d_out_idx, batch_size * k * sizeof(uint32_t), hipMemcpyDeviceToHost));

    // 释放显存
    CUDA_CHECK(hipFree(d_query));
    CUDA_CHECK(hipFree(d_new_base));
    CUDA_CHECK(hipFree(d_new_to_old));
    CUDA_CHECK(hipFree(d_cluster_start));
    CUDA_CHECK(hipFree(d_selected_clusters));
    CUDA_CHECK(hipFree(d_out_dist));
    CUDA_CHECK(hipFree(d_out_idx));

    // 构建结果
    std::vector<std::priority_queue<std::pair<float, uint32_t>>> results(batch_size);
    for (size_t i = 0; i < batch_size; ++i) {
        for (size_t j = 0; j < k; ++j) {
            size_t idx = i * k + j;
            results[i].emplace(host_dist[idx], host_idx[idx]);
        }
    }
    return results;
}

